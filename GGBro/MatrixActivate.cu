#include "hip/hip_runtime.h"
// incrementArray.cu
// Utilities and system includes
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
//-----------------------------------------------------------------------------------

void NN_OnHost(float *activity, float *weights, int N)
{
	int i, j;
	float new_activity[N];
	for (i = 0; i<N; i++) {
		new_activity[i] = 0;
		for (j = 0; j<N; j++) {
			new_activity[i] += activity[j] * weights[(j*N) + i];
		}
	}
	for (i = 0; i < N; i++) {
		activity[i] = 1 / (1 + exp(-new_activity[i]));
	}
}
//-----------------------------------------------------------------------------------

__global__ void NN_OnDevice(float *activity, float *weights, float *new_activity, int N)
{
	int j, idx = threadIdx.x;
	new_activity[idx] = 0;
	for (j = 0; j<N; j++) {
		new_activity[idx] += activity[j] * weights[(j*N) + idx];
	}
	__syncthreads();
	activity[idx] = 1 / (1 + exp(-new_activity[idx]));
}
//-----------------------------------------------------------------------------------

int main(void)
{
	hipSetDevice(0);
	float *activity_h, *weights_h, *new_activity_h;		   // pointers to host memory
	float *activity_d, *weights_d, *new_activity_d;		   // pointer to device memory
	int i, j, N = 100;
	size_t size = N * sizeof(float);

	//timer stuff
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate arrays on host
	activity_h = (float *)malloc(size);
	new_activity_h = (float *)malloc(size);
	weights_h = (float *)malloc(size*size);

	// allocate array on device 
	hipMalloc((void **)&activity_d, size);
	hipMalloc((void **)&new_activity_d, size);
	hipMalloc((void **)&weights_d, size*size);

	// initialization of host data
	for (i = 0; i<N; i++) {
		activity_h[i] = (float(rand() % 100) / 100);
		for (j = 0; j<N; j++) {
			weights_h[(j*N) + i] = (float(rand() % 200) / 100) - 1;
			//printf("%f ",weights_h[(j*N)+i]);
		}
		//printf("%f ",activity_h[i]);
	}
	//printf("\n");

	// copy data from host to device
	hipMemcpy(activity_d, activity_h, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(weights_d, weights_h, sizeof(float)*N*N, hipMemcpyHostToDevice);

	// do calculation on host
	NN_OnHost(activity_h, weights_h, N);

	for (i = 0; i<10; i++) printf("%f ", activity_h[i]);
	printf("\n");

	//start timer 
	hipEventRecord(start, 0);

	// do calculation on device:	
	NN_OnDevice <<< 1, N >>> (activity_d, weights_d, new_activity_d, N);

	// block until the device has completed
	hipDeviceSynchronize();

	//end timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	// Retrieve result from device and store in b_h
	hipMemcpy(new_activity_h, activity_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	for (i = 0; i<10; i++) printf("%f ", new_activity_h[i]);
	printf("\n");
	printf("time = %f\n\n", elapsedTime);

	// cleanup
	free(activity_h); free(weights_h); free(new_activity_h);
	hipFree(activity_d); hipFree(weights_d); hipFree(new_activity_d);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}